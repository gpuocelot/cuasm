#include "hip/hip_runtime.h"


// compile: nvcc -o main main.cu  -lcublas -lcudart -lcuda -lcurand
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>

#include <iostream>
#include <stdio.h>
#include <string.h>

// Must be multiples of 16 for wmma code to work
#define MATRIX_M 16384
#define MATRIX_N 16384
#define MATRIX_K 16384

// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// Define some error checking macros.
#define cudaErrCheck(stat)                         \
    {                                              \
        cudaErrCheck_((stat), __FILE__, __LINE__); \
    }
void cudaErrCheck_(hipError_t stat, const char* file, int line)
{
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

#define curandErrCheck(stat)                         \
    {                                                \
        curandErrCheck_((stat), __FILE__, __LINE__); \
    }

void curandErrCheck_(hiprandStatus_t stat, const char* file, int line)
{
    if (stat != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
    }
}


__global__ void convertFp32ToFp16(half* out, float* in, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx];
    }
}

char* concat(const char *s1, const char *s2)
{
    char *result = (char*)malloc(strlen(s1) + strlen(s2) + 1); // +1 for the null-terminator
    // in real code you would check for errors in malloc here
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void bench() {
}

void run(const char * fn, const char* func){
        // load module
        char * file_name = concat(fn, ".cubin");
	hipError_t cuResult;
    hipCtx_t cuContext;


        hipModule_t module;
        hipFunction_t kernel;

    // Initialize CUDA context
    cuResult = hipInit(0);
    if (cuResult != hipSuccess) {
        std::cerr << "hipInit failed with error code: " << cuResult << std::endl;
        return ;
    }

    // Create CUDA context
    cuResult = hipCtxCreate(&cuContext, 0, 0);
    if (cuResult != hipSuccess) {
        std::cerr << "hipCtxCreate failed with error code: " << cuResult << std::endl;
        return ;
    }

        cuResult = hipModuleLoad(&module, file_name);

    if (cuResult != hipSuccess) {
        std::cerr << "hipModuleLoad failed with error code: " << cuResult << std::endl;
        // Handle the failure appropriately
        return ;
    }
        cuResult = hipModuleGetFunction(&kernel, module, func);
    if (cuResult != hipSuccess) {
        std::cerr << "hipModuleGetFunction failed with error code: " << cuResult << std::endl;
        // Handle the failure appropriately, maybe cleanup
        hipModuleUnload(module); // Unload the module in case of failure
        return ;
    }

        // args
        float* a_fp32;
        float* b_fp32;
        half* a_fp16;
        half* b_fp16;

        float* c;
        float* c_wmma;

        float* c_host_wmma;

        hiprandGenerator_t gen;

        hipEvent_t startWMMA;
        hipEvent_t stopWMMA;

        cudaErrCheck(hipEventCreate(&startWMMA));
        cudaErrCheck(hipEventCreate(&stopWMMA));

        cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));

        cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));

        c_host_wmma = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

        curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

        curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
        curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

        // hiprand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        convertFp32ToFp16<<<(MATRIX_M * MATRIX_K + 255) / 256, 256>>>(a_fp16, a_fp32, MATRIX_M * MATRIX_K);
        convertFp32ToFp16<<<(MATRIX_K * MATRIX_N + 255) / 256, 256>>>(b_fp16, b_fp32, MATRIX_K * MATRIX_N);

        curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));

        curandErrCheck(hiprandDestroyGenerator(gen));

        cudaErrCheck(hipMemcpy(c_wmma, c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToDevice));

	// int *output;
	// hipMalloc((void**)&output, sizeof(int)*128);


	// void * args[1] = {&output};

	// hipModuleLaunchKernel(kernel, 1, 1, 1,
	// 		32, 1, 1,
	// 		32*1024, 0, args, 0);

	// Set arguments
	int M =  MATRIX_M;
	int N =  MATRIX_N;
	int K =  MATRIX_K;
        float alpha = 2.0f;
        float beta = 2.0f;

        printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

	void* args[] = { &a_fp16, &b_fp16, &c_wmma, &M, &N, &K, &alpha, &beta };

        // kernel args
        dim3 gridDim;
        dim3 blockDim;

        // blockDim.x must be a multple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
        gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

        printf("gridDim %d, %d\n", gridDim.x, gridDim.y);  // 256, 256
        printf("blockDim %d, %d\n", blockDim.x, blockDim.y);  // 128, 4


	// Launch the kernel
	cudaErrCheck(hipEventRecord(startWMMA));
	hipModuleLaunchKernel(
	kernel,
	gridDim.x, gridDim.y, gridDim.z,
	blockDim.x, blockDim.y, blockDim.z,
	0, /* Shared memory size - if using shared memory */
	NULL, /* Stream identifier - if using streams */
	args, /* Kernel arguments */
	NULL /* Extra options */
	);
        cudaErrCheck(hipEventRecord(stopWMMA));
        cudaErrCheck(hipEventSynchronize(stopWMMA));

	// TODO benchmark
        float milliseconds = 0.0f;

    cudaErrCheck(hipEventElapsedTime(&milliseconds, startWMMA, stopWMMA));

    std::cout << "Time taken: " << milliseconds << " milliseconds" << std::endl;

    cudaErrCheck(hipEventDestroy(startWMMA));
    cudaErrCheck(hipEventDestroy(stopWMMA));

}



int main(){
	run("out", "_Z12wmma_exampleP6__halfS0_Pfiiiff");
	return 0;
}
