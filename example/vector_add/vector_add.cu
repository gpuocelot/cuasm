
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1000; // Number of elements in the vectors
    size_t bytes = n * sizeof(float);

    // Allocate memory on the host
    float *h_a, *h_b, *h_c;
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    // Initialize input vectors
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Allocate memory on the device
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy vectors from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Set grid and block dimensions
    int blockSize = 256;
    int gridSize = (int)ceil((float)n / blockSize);

    // Launch the kernel
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy result from device to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < 10; i++) {
        printf("%.2f + %.2f = %.2f\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free memory on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free memory on the host
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}